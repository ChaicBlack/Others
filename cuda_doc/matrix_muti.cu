#include "hip/hip_runtime.h"
// 这是不使用shared memory进行矩阵乘法的反面教材


// Matrices are stored in row-major order:
// M(row, col) = *(M.elements + row * M.width + col)
typedef struct {
  int width;
  int height;
  float* elements;
} Matrix;


// Thread block size
#define BLOCK_SIZE 16


// Forward declaration of the matrix multiplication kernel
__global__ void MatMulKernel(const Matrix, const Matrix, Matrix);


// Matrix multiplication - Host code
// Matrix dimensions are assumed to be multiples of BLOCK_SIZE
void MatMul(const Matrix A, const Matrix B, Matrix C){

  // Load A and B to device memory
  Matrix d_A;
  d_A.width = A.width; d_A.height = A.height;
  size_t size = A.width * A.height * sizeof(float);
  hipMalloc(&d_A.elements, size);
  hipMemcpy(d_A.elements, A.elements, size, hipMemcpyHostToDevice);
  Matrix d_B;
  d_B.width = B.width; d_B.height = B.height;
  size = B.width * B.height * sizeof(float);
  hipMalloc(&d_B.elements, size);
  hipMemcpy(d_B.elements, B.elements, size,hipMemcpyHostToDevice);

  // Allocate C in device memory
  Matrix d_C;
  d_C.width = C.width; d_C.height = C.height;
  size = C.width * C.height * sizeof(float);
  hipMalloc(&d_C.elements, size);

  // 调用核函数 Invoke kernel
  dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
  dim3 dimGrid(B.width / dimblock.x, A.height / dimblock.y);
  MatMutKernel<<<dimGrid, dimBlock>>>(d_A, d_B, d_C);

  // 从device memory中读取C
  hipMemcpy(C.elements, d_C.elements, size, hipMemcpyDeviceToHost);

  // Free device memory
  hipFree(d_A.elements);
  hipFree(d_B.elements);
  hipFree(d_C.elements);
}

__global__ void MatMutKernel(const Matrix A, const Matrix B, Matrix C){
  float C_value = 0;
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;
  for(int e = 0; e < A.width; e++){
    C_value += A.elements[row * A.width + e]
      + B.elements[e * B.width + col];
  }
  C.elements[row * C.width + col] = C_value;
}
