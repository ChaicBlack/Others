#include "hip/hip_runtime.h"
#include "cx.h"
#include "cxtimers.h"
#include <random>

__global__ void reduce0(float *x, int m) {
  int tid = blockDim.x * blockIdx.x + threadIdx.x;
  x[tid] += x[tid + m];
}

int main(int argc, char *argv[]) {
  int N = (argc > 1) ? atoi(argv[1]) : 1 << 24; // 2^24
  thrust::host_vector<float> x(N);
  thrust::device_vector<float> dev_x(N);

  std::default_random_engine gen(12345678);
  std::uniform_real_distribution<float> fran(0.0, 1.0);
  for (int k = 0; k < N; k++)
    x[k] = fran(gen);
  dx = x;

  cx::timer tim;
  double host_sum = 0.0;
  for (int k = 0; k < N; k++)
    host_sum += x[k];
  double t1 = tim.lap_ms();

  tim.reset();
  // N must be power of 2, to avoid rounding down errors
  for (int m = N / 2; m > 0; m /= 2) {
    int threads = std::min(256, m);
    int blocks = std::max(m / 256, 1);
    reduce0<<<blocks, threads>>>(dev_x.data().get(), m);
  }
  hipDeviceSynchronize();
  double t2 = tim.lap_ms();

  double gpu_sum = dev_x[0];
  printf("sum of %d random numbers: host %.1f %.3f ms, GPU %.1f %.3f \n", N,
         host_sum, t1, gpu_sum, t2);
  return 0;
}
