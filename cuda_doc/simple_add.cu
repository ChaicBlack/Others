#include "hip/hip_runtime.h"
#include "../commen/book.h"

#define N (33 * 1024)

__global__ void add(int * a, int * b, int * c){
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  while(tid < N){
    c[tid] = a[tid] + b[tid];
    tid += blockDim.x * gridDim.x;
  }
}

int main(){
  int a[N], b[N], c[N];
  int *dev_a, *dev_b, *dev_c;

  hipMalloc((void**)&dev_a, N * sizeof(int));
  hipMalloc((void**)&dev_b, N * sizeof(int));
  hipMalloc((void**)&dev_c, N * sizeof(int));

  for(int i = 0; i < N; i++){
    a[i] = i;
    b[i] = N - i;
  }

  hipMemcpy(dev_a, a, N * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(dev_b, b, N * sizeof(int), hipMemcpyHostToDevice);

  add<<<128, 128>>>(dev_a, dev_b, dev_c);

  hipMemcpy(c, dev_c, N * sizeof(int), hipMemcpyDeviceToHost);

  hipFree(dev_a);
  hipFree(dev_b);
  hipFree(dev_c);
}
